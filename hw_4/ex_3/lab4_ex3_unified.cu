
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

//#define DataType double
#define DataType float
#define DEBUG 0

// Compute C = A * B (on CPU)
void matmul(DataType *A, DataType *B, DataType *C, int numARows,
            int numAColumns, int numBRows, int numBColumns) {
    // Multiplying matrices A and B and storing it in C
    for (int i = 0; i < numARows; ++i) {
        for (int j = 0; j < numBColumns; ++j) {
            C[i*numBColumns + j] = 0.0;
            for (int k = 0; k < numAColumns; ++k) {
                C[i*numBColumns + j] += A[i*numAColumns + k] * B[k*numBColumns + j];
            }
            #if DEBUG
            printf("C[%d, %d] = %f\n", i, j, C[i*numBColumns + j]);
            #endif
        }
    }
}

// Compute C = A * B (on GPU)
__global__ void gemm(DataType *A, DataType *B, DataType *C, int numARows,
                      int numAColumns, int numBRows, int numBColumns){
    //@@ Insert code to implement matrix multiplication here
    // x corresponds to the number of columns, y corresponds to the number of rows
    const int col = blockIdx.x * blockDim.x + threadIdx.x;
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    if ((col >= numBColumns) || (row >= numARows)) return;

    DataType tmpSum = 0.0;
    for (int k = 0; k < numAColumns; ++k) {
        tmpSum += A[row*numAColumns + k] * B[k*numBColumns + col];
    }
    C[row*numBColumns + col] = tmpSum;
    #if DEBUG
    printf("C[%d, %d] = %f\n", row, col, C[row*numBColumns + col]);
    #endif
}

int main(int argc, char **argv) {
    DataType *umA; // The A matrix
    DataType *umB; // The B matrix
    DataType *umC; // The output C matrix
    DataType *resultRef; // The reference result
    int numARows;    // number of rows in the matrix A
    int numAColumns; // number of columns in the matrix A
    int numBRows;    // number of rows in the matrix B
    int numBColumns; // number of columns in the matrix B
    int numCRows;
    int numCColumns;

    //@@ Insert code below to read in numARows, numAColumns, numBColumns from args
    if (argc != 5) {
        printf("ERROR: Exactly four input parameters are required to run the program (%d != 4).\n", argc);
        exit(1);
    }
    numARows = atoi(argv[1]);
    numAColumns = atoi(argv[2]);
    numBRows = atoi(argv[3]);
    numBColumns = atoi(argv[4]);
    numCRows = numARows;
    numCColumns = numBColumns;
    printf("Input matrix dim (%d x %d) (%d x %d) (%d x %d)\n", numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);
    if (numAColumns != numBRows) {
        printf("ERROR: Matrix A must have the same number of columns as the number of rows of matrix B (%d != %d).\n", numAColumns, numBRows);
        return 0;
    }

    //@@ Insert code below to allocate unified memory memory for input and output
    hipMallocManaged(&umA, numARows * numAColumns * sizeof(DataType)); // umA[i*numAColumns + j] is equivalent to the usual umA[i][j]
    hipMallocManaged(&umB, numBRows * numBColumns * sizeof(DataType)); // umB[i*numBColumns + j] is equivalent to the usual umB[i][j]
    hipMallocManaged(&umC, numCRows * numCColumns * sizeof(DataType)); // umC[i*numCColumns + j] is equivalent to the usual umC[i][j]
    hipMallocManaged(&resultRef, numCRows * numCColumns * sizeof(DataType));

    //@@ Insert code below to initialize umA and umB to random numbers, and create reference result in CPU
    for (int i = 0; i < numARows; ++i) {
        for (int j = 0; j < numAColumns; ++j) {
            DataType randomNumber = rand() / (DataType) RAND_MAX; // Random number in interval [0, 1.0]
            umA[i*numAColumns + j] = randomNumber;
            #if DEBUG
            printf("umA[%d, %d] = %f\n", i, j, umA[i*numBColumns + j]);
            #endif
        }
    }
    for (int i = 0; i < numBRows; ++i) {
        for (int j = 0; j < numBColumns; ++j) {
            DataType randomNumber = rand() / (DataType) RAND_MAX; // Random number in interval [0, 1.0]
            umB[i*numBColumns + j] = randomNumber;
            #if DEBUG
            printf("umB[%d, %d] = %f\n", i, j, umB[i*numBColumns + j]);
            #endif
        }
    }
    // Calculate reference result
    matmul(umA, umB, resultRef, numARows, numAColumns, numBRows, numBColumns);

    //@@ Initialize the grid and block dimensions here
    // x corresponds to the number of columns, y corresponds to the number of rows
    int Dbx = 16;
    int Dby = 16;
    int Dgx = (numCColumns + Dbx - 1) / Dbx;
    int Dgy = (numCRows + Dby - 1) / Dby;

    //@@ Launch the GPU Kernel here
    gemm <<<dim3(Dgx, Dgy, 1), dim3(Dbx, Dby, 1)>>>(umA, umB, umC, numARows, numAColumns, numBRows, numBColumns);

    // Explicitly synchronize on the host side before directly accessing the output of the kernel
    hipDeviceSynchronize();

    //@@ Insert code below to compare the output with the reference
    int equality = 1;
    for (int i = 0; i < numCRows; ++i) {
        for (int j = 0; j < numCColumns; ++j) {
            if (fabs(umC[i*numCColumns + j] - resultRef[i*numCColumns + j]) > 1e-4) { // Compare if elements are approximately equal
                equality = 0;
                #if DEBUG
                printf("Position: [%d, %d], Difference: %f\n", i, j, fabs(umC[i*numCColumns + j] - resultRef[i*numCColumns + j]));
                #endif
                break;
            }
        }
    }
    if (equality == 1) {
        printf("CPU and GPU results are equal.\n");
    } else {
        printf("CPU and GPU results are NOT equal.\n");
    }

    //@@ Free the unified memory here
    hipFree(umA);
    hipFree(umB);
    hipFree(umC);
    hipFree(resultRef);

    return 0;
}